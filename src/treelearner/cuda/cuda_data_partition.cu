
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA

#include "cuda_data_partition.hpp"
#include <LightGBM/tree.h>

namespace LightGBM {

#define CONFLICT_FREE_INDEX(n) \
  ((n) + ((n) >> LOG_NUM_BANKS_DATA_PARTITION)) \

__device__ void PrefixSum(uint32_t* elements, unsigned int n) {
  unsigned int offset = 1;
  unsigned int threadIdx_x = threadIdx.x;
  const unsigned int conflict_free_n_minus_1 = CONFLICT_FREE_INDEX(n - 1);
  const uint32_t last_element = elements[conflict_free_n_minus_1];
  __syncthreads();
  for (int d = (n >> 1); d > 0; d >>= 1) {
    if (threadIdx_x < d) {
      const unsigned int src_pos = offset * (2 * threadIdx_x + 1) - 1;
      const unsigned int dst_pos = offset * (2 * threadIdx_x + 2) - 1;
      elements[CONFLICT_FREE_INDEX(dst_pos)] += elements[CONFLICT_FREE_INDEX(src_pos)];
    }
    offset <<= 1;
    __syncthreads();
  }
  if (threadIdx_x == 0) {
    elements[conflict_free_n_minus_1] = 0; 
  }
  __syncthreads();
  for (int d = 1; d < n; d <<= 1) {
    offset >>= 1;
    if (threadIdx_x < d) {
      const unsigned int dst_pos = offset * (2 * threadIdx_x + 2) - 1;
      const unsigned int src_pos = offset * (2 * threadIdx_x + 1) - 1;
      const unsigned int conflict_free_dst_pos = CONFLICT_FREE_INDEX(dst_pos);
      const unsigned int conflict_free_src_pos = CONFLICT_FREE_INDEX(src_pos);
      const uint32_t src_val = elements[conflict_free_src_pos];
      elements[conflict_free_src_pos] = elements[conflict_free_dst_pos];
      elements[conflict_free_dst_pos] += src_val;
    }
    __syncthreads();
  }
  if (threadIdx_x == 0) {
    elements[CONFLICT_FREE_INDEX(n)] = elements[conflict_free_n_minus_1] + last_element;
  }
}

__global__ void FillDataIndicesBeforeTrainKernel(const data_size_t* cuda_num_data,
  data_size_t* data_indices) {
  const data_size_t num_data_ref = *cuda_num_data;
  const unsigned int data_index = threadIdx.x + blockIdx.x * blockDim.x;
  if (data_index < num_data_ref) {
    data_indices[data_index] = data_index;
  }
}

void CUDADataPartition::LaunchFillDataIndicesBeforeTrain() {
  const int num_blocks = (num_data_ + FILL_INDICES_BLOCK_SIZE_DATA_PARTITION - 1) / FILL_INDICES_BLOCK_SIZE_DATA_PARTITION;
  FillDataIndicesBeforeTrainKernel<<<num_blocks, FILL_INDICES_BLOCK_SIZE_DATA_PARTITION>>>(cuda_num_data_, cuda_data_indices_); 
}

__global__ void GenDataToLeftBitVectorKernel(const int* leaf_index, const data_size_t* cuda_leaf_data_start,
  const data_size_t* cuda_leaf_num_data, const data_size_t* cuda_data_indices, const int* best_split_feature,
  const uint32_t* best_split_threshold, const int* cuda_num_features, const uint8_t* cuda_data,
  const uint32_t* default_bin, const uint32_t* most_freq_bin, const uint8_t* default_left,
  const uint32_t* min_bin, const uint32_t* max_bin, const uint8_t* missing_is_zero, const uint8_t* missing_is_na,
  const uint8_t* mfb_is_zero, const uint8_t* mfb_is_na,
  uint8_t* cuda_data_to_left) {
  /*if (blockIdx.x == 0 && threadIdx.x == 0) {
    printf("GenDataToLeftBitVectorKernel step 0\n");
  }*/
  const int leaf_index_ref = *leaf_index;
  /*if (blockIdx.x == 0 && threadIdx.x == 0) {
    printf("GenDataToLeftBitVectorKernel leaf_index_ref = %d\n", leaf_index_ref);
  }*/
  const int best_split_feature_ref = best_split_feature[leaf_index_ref];
  const int num_features_ref = *cuda_num_features;
  const uint32_t best_split_threshold_ref = best_split_threshold[leaf_index_ref];
  const uint8_t default_left_ref = default_left[leaf_index_ref];
  const data_size_t leaf_num_data_offset = cuda_leaf_data_start[leaf_index_ref];
  const data_size_t num_data_in_leaf = cuda_leaf_num_data[leaf_index_ref];
  const data_size_t* data_indices_in_leaf = cuda_data_indices + leaf_num_data_offset;
  const unsigned int local_data_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (local_data_index < num_data_in_leaf) {
    const unsigned int global_data_index = data_indices_in_leaf[local_data_index];
    const unsigned int global_feature_value_index = global_data_index * num_features_ref + best_split_feature_ref;
    const uint32_t default_bin_ref = default_bin[best_split_feature_ref];
    const uint32_t most_freq_bin_ref = most_freq_bin[best_split_feature_ref];
    const uint32_t max_bin_ref = max_bin[best_split_feature_ref];
    const uint32_t min_bin_ref = min_bin[best_split_feature_ref];
    const uint8_t missing_is_zero_ref = missing_is_zero[best_split_feature_ref];
    const uint8_t missing_is_na_ref = missing_is_na[best_split_feature_ref];
    const uint8_t mfb_is_zero_ref = mfb_is_zero[best_split_feature_ref];
    const uint8_t mfb_is_na_ref = mfb_is_na[best_split_feature_ref];
    /*if (blockIdx.x == 0 && threadIdx.x == 0) {
      printf("GenDataToLeftBitVectorKernel step 1\n");
    }*/
    uint32_t th = best_split_threshold_ref + min_bin_ref;
    uint32_t t_zero_bin = min_bin_ref + default_bin_ref;
    if (most_freq_bin_ref == 0) {
      --th;
      --t_zero_bin;
    }
    uint8_t split_default_to_left = 0;
    uint8_t split_missing_default_to_left = 0;
    if (most_freq_bin_ref <= best_split_threshold_ref) {
      split_default_to_left = 1;
    }
    if (missing_is_zero_ref || missing_is_na_ref) {
      if (default_left_ref) {
        split_missing_default_to_left = 1;
      }
    }
    /*if (blockIdx.x == 0 && threadIdx.x == 0) {
      printf("GenDataToLeftBitVectorKernel step 2\n");
    }*/
    if (local_data_index < static_cast<unsigned int>(num_data_in_leaf)) {
      /*if (blockIdx.x == 0 && threadIdx.x == 0) {
        printf("GenDataToLeftBitVectorKernel step 3\n");
      }*/
      const uint32_t bin = static_cast<uint32_t>(cuda_data[global_feature_value_index]);
      if (min_bin_ref < max_bin_ref) {
        if ((missing_is_zero_ref && !mfb_is_zero_ref && bin == t_zero_bin)) {
          cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        } else if (bin < min_bin_ref || bin > max_bin_ref) {
          if ((missing_is_na_ref || mfb_is_na_ref) || (missing_is_zero_ref || mfb_is_zero_ref)) {
            cuda_data_to_left[local_data_index] = split_missing_default_to_left;
          } else {
            cuda_data_to_left[local_data_index] = split_default_to_left;
          }
        } else if (bin > th) {
          cuda_data_to_left[local_data_index] = 0;
        } else {
          cuda_data_to_left[local_data_index] = 1;
        }
      } else {
        if (missing_is_zero_ref || !mfb_is_zero_ref && bin == t_zero_bin) {
          cuda_data_to_left[local_data_index] = split_missing_default_to_left;
        } else if (bin != max_bin_ref) {
          if ((missing_is_na_ref && mfb_is_na_ref) || (missing_is_zero_ref && mfb_is_zero_ref)) {
            cuda_data_to_left[local_data_index] = split_missing_default_to_left;
          } else {
            cuda_data_to_left[local_data_index] = split_default_to_left;
          }
        } else {
          if (missing_is_na_ref && !mfb_is_na_ref) {
            cuda_data_to_left[local_data_index] = split_missing_default_to_left;
          } else {
            cuda_data_to_left[local_data_index] = split_default_to_left;
          }
        }
      }
      /*if (blockIdx.x == 0 && threadIdx.x == 0) {
        printf("GenDataToLeftBitVectorKernel step 4\n");
      }*/
    }
  }
}

void CUDADataPartition::LaunchGenDataToLeftBitVectorKernel(const int* leaf_index, const data_size_t num_data_in_leaf, const int* best_split_feature,
  const uint32_t* best_split_threshold, const uint8_t* best_split_default_left) {
  const int num_blocks = std::max(80, (num_data_in_leaf + SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION - 1) / SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION);
  int split_indices_block_size_data_partition = (num_data_in_leaf + num_blocks - 1) / num_blocks - 1;
  int split_indices_block_size_data_partition_aligned = 1;
  while (split_indices_block_size_data_partition > 0) {
    split_indices_block_size_data_partition_aligned <<= 1;
    split_indices_block_size_data_partition >>= 1;
  }
  GenDataToLeftBitVectorKernel<<<num_blocks, split_indices_block_size_data_partition_aligned>>>(
    leaf_index, cuda_leaf_data_start_, cuda_leaf_num_data_,
    cuda_data_indices_, best_split_feature, best_split_threshold,
    cuda_num_features_, cuda_data_,
    cuda_feature_default_bins_, cuda_feature_most_freq_bins_, best_split_default_left,
    cuda_feature_min_bins_, cuda_feature_max_bins_, cuda_feature_missing_is_zero_, cuda_feature_missing_is_na_,
    cuda_feature_mfb_is_zero_, cuda_feature_mfb_is_na_,
    cuda_data_to_left_);
  SynchronizeCUDADevice();
}

__global__ void PrepareOffsetKernel(const int* leaf_index,
  const data_size_t* cuda_leaf_num_data,  const uint8_t* split_to_left_bit_vector,
  data_size_t* block_to_left_offset_buffer, data_size_t* block_to_right_offset_buffer,
  const int split_indices_block_size_data_partition) {
  const unsigned int blockDim_x = blockDim.x;
  __shared__ uint32_t thread_to_left_offset_cnt[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 1 +
    (SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 1) / NUM_BANKS_DATA_PARTITION];
  //__shared__ uint32_t thread_to_right_offset_cnt[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 1 +
  //  (SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 1) / NUM_BANKS_DATA_PARTITION];
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int conflict_free_threadIdx_x = CONFLICT_FREE_INDEX(threadIdx_x);
  const unsigned int global_read_index = blockIdx.x * blockDim.x * 2 + threadIdx_x;
  const data_size_t num_data_in_leaf_ref = cuda_leaf_num_data[*leaf_index];
  /*if (blockIdx.x == 0 && threadIdx_x == 0) {
    printf("PrepareOffsetKernel leaf_index = %d, num_data_in_leaf = %d\n", (*leaf_index), num_data_in_leaf_ref);
  }*/
  if (global_read_index < num_data_in_leaf_ref) {
    const uint8_t bit = split_to_left_bit_vector[global_read_index];
    thread_to_left_offset_cnt[conflict_free_threadIdx_x] = bit;
    //thread_to_right_offset_cnt[conflict_free_threadIdx_x] = 1 - bit;
  } else {
    thread_to_left_offset_cnt[conflict_free_threadIdx_x] = 0;
    //thread_to_right_offset_cnt[conflict_free_threadIdx_x] = 0;
  }
  const unsigned int conflict_free_threadIdx_x_offseted = CONFLICT_FREE_INDEX(threadIdx_x + blockDim_x);
  if (global_read_index + blockDim_x < num_data_in_leaf_ref) {
    const uint8_t bit = split_to_left_bit_vector[global_read_index + blockDim_x];
    thread_to_left_offset_cnt[conflict_free_threadIdx_x_offseted] = bit;
    //thread_to_right_offset_cnt[conflict_free_threadIdx_x_offseted] = 1 - bit;
  } else {
    thread_to_left_offset_cnt[conflict_free_threadIdx_x_offseted] = 0;
    //thread_to_right_offset_cnt[conflict_free_threadIdx_x_offseted] = 0;
  }
  __syncthreads();
  PrefixSum(thread_to_left_offset_cnt, split_indices_block_size_data_partition);
  //PrefixSum(thread_to_right_offset_cnt, split_indices_block_size_data_partition);
  __syncthreads();
  if (threadIdx_x == 0) {
    const unsigned int conflict_free_blockDim_x_times_2 = CONFLICT_FREE_INDEX(blockDim_x << 1);
    const data_size_t num_data_in_block = (blockIdx.x + 1) * blockDim.x * 2 <= num_data_in_leaf_ref ? static_cast<data_size_t>(blockDim_x * 2) :
      num_data_in_leaf_ref - static_cast<data_size_t>(blockIdx.x * blockDim.x * 2);
    if (num_data_in_block > 0) {
      const data_size_t data_to_left = static_cast<data_size_t>(thread_to_left_offset_cnt[conflict_free_blockDim_x_times_2]);
      block_to_left_offset_buffer[blockIdx.x + 1] = data_to_left;
      block_to_right_offset_buffer[blockIdx.x + 1] = num_data_in_block - data_to_left;
    } else {
      block_to_left_offset_buffer[blockIdx.x + 1] = 0;
      block_to_right_offset_buffer[blockIdx.x + 1] = 0;
    }
  }
}

__global__ void AggregateBlockOffsetKernel(const int* leaf_index, data_size_t* block_to_left_offset_buffer,
  data_size_t* block_to_right_offset_buffer, data_size_t* cuda_leaf_data_start,
  data_size_t* cuda_leaf_data_end, data_size_t* cuda_leaf_num_data, const data_size_t* cuda_data_indices,
  int* cuda_cur_num_leaves,
  const double* best_left_sum_gradients, const double* best_left_sum_hessians, const data_size_t* best_left_count,
  const double* best_left_gain, const double* best_left_leaf_value,
  const double* best_right_sum_gradients, const double* best_right_sum_hessians, const data_size_t* best_right_count,
  const double* best_right_gain, const double* best_right_leaf_value,
  // for leaf splits information update
  int* smaller_leaf_cuda_leaf_index_pointer, double* smaller_leaf_cuda_sum_of_gradients_pointer,
  double* smaller_leaf_cuda_sum_of_hessians_pointer, data_size_t* smaller_leaf_cuda_num_data_in_leaf_pointer,
  double* smaller_leaf_cuda_gain_pointer, double* smaller_leaf_cuda_leaf_value_pointer,
  const data_size_t** smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer,
  hist_t** smaller_leaf_cuda_hist_pointer_pointer,
  int* larger_leaf_cuda_leaf_index_pointer, double* larger_leaf_cuda_sum_of_gradients_pointer,
  double* larger_leaf_cuda_sum_of_hessians_pointer, data_size_t* larger_leaf_cuda_num_data_in_leaf_pointer,
  double* larger_leaf_cuda_gain_pointer, double* larger_leaf_cuda_leaf_value_pointer,
  const data_size_t** larger_leaf_cuda_data_indices_in_leaf_pointer_pointer,
  hist_t** larger_leaf_cuda_hist_pointer_pointer,
  const int* cuda_num_total_bin,
  hist_t* cuda_hist, hist_t** cuda_hist_pool, const int split_indices_block_size_data_partition) {
  __shared__ uint32_t block_to_left_offset[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 2 +
    (SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 2) / NUM_BANKS_DATA_PARTITION];
  __shared__ uint32_t block_to_right_offset[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 2 +
    (SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 2) / NUM_BANKS_DATA_PARTITION];
  const int leaf_index_ref = *leaf_index;
  const data_size_t num_data_in_leaf = cuda_leaf_num_data[leaf_index_ref];
  const unsigned int blockDim_x = blockDim.x;
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int conflict_free_threadIdx_x = CONFLICT_FREE_INDEX(threadIdx_x);
  const unsigned int conflict_free_threadIdx_x_plus_blockDim_x = CONFLICT_FREE_INDEX(threadIdx_x + blockDim_x);
  const uint32_t num_blocks = (num_data_in_leaf + split_indices_block_size_data_partition - 1) / split_indices_block_size_data_partition;
  const uint32_t num_aggregate_blocks = (num_blocks + split_indices_block_size_data_partition - 1) / split_indices_block_size_data_partition;
  uint32_t left_prev_sum = 0;
  for (uint32_t block_id = 0; block_id < num_aggregate_blocks; ++block_id) {
    const unsigned int read_index = block_id * blockDim_x * 2 + threadIdx_x;
    if (read_index < num_blocks) {
      block_to_left_offset[conflict_free_threadIdx_x] = block_to_left_offset_buffer[read_index + 1];
    } else {
      block_to_left_offset[conflict_free_threadIdx_x] = 0;
    }
    const unsigned int read_index_plus_blockDim_x = read_index + blockDim_x;
    if (read_index_plus_blockDim_x < num_blocks) {
      block_to_left_offset[conflict_free_threadIdx_x_plus_blockDim_x] = block_to_left_offset_buffer[read_index_plus_blockDim_x + 1];
    } else {
      block_to_left_offset[conflict_free_threadIdx_x_plus_blockDim_x] = 0;
    }
    if (threadIdx_x == 0) {
      block_to_left_offset[0] += left_prev_sum;
    }
    __syncthreads();
    PrefixSum(block_to_left_offset, split_indices_block_size_data_partition);
    __syncthreads();
    if (threadIdx_x == 0) {
      left_prev_sum = block_to_left_offset[CONFLICT_FREE_INDEX(split_indices_block_size_data_partition)];
    }
    if (read_index < num_blocks) {
      const unsigned int conflict_free_threadIdx_x_plus_1 = CONFLICT_FREE_INDEX(threadIdx_x + 1);
      block_to_left_offset_buffer[read_index + 1] = block_to_left_offset[conflict_free_threadIdx_x_plus_1];
    }
    if (read_index_plus_blockDim_x < num_blocks) {
      const unsigned int conflict_free_threadIdx_x_plus_1_plus_blockDim_x = CONFLICT_FREE_INDEX(threadIdx_x + 1 + blockDim_x);
      block_to_left_offset_buffer[read_index_plus_blockDim_x + 1] = block_to_left_offset[conflict_free_threadIdx_x_plus_1_plus_blockDim_x];
    }
    __syncthreads();
  }
  const unsigned int to_left_total_cnt = block_to_left_offset_buffer[num_blocks];
  uint32_t right_prev_sum = to_left_total_cnt;
  for (uint32_t block_id = 0; block_id < num_aggregate_blocks; ++block_id) {
    const unsigned int read_index = block_id * blockDim_x * 2 + threadIdx_x;
    if (read_index < num_blocks) {
      block_to_right_offset[conflict_free_threadIdx_x] = block_to_right_offset_buffer[read_index + 1];
    } else {
      block_to_right_offset[conflict_free_threadIdx_x] = 0;
    }
    const unsigned int read_index_plus_blockDim_x = read_index + blockDim_x;
    if (read_index_plus_blockDim_x < num_blocks) {
      block_to_right_offset[conflict_free_threadIdx_x_plus_blockDim_x] = block_to_right_offset_buffer[read_index_plus_blockDim_x + 1];
    } else {
      block_to_right_offset[conflict_free_threadIdx_x_plus_blockDim_x] = 0;
    }
    if (threadIdx_x == 0) {
      block_to_right_offset[0] += right_prev_sum;
    }
    __syncthreads();
    PrefixSum(block_to_right_offset, split_indices_block_size_data_partition);
    __syncthreads();
    if (threadIdx_x == 0) {
      right_prev_sum = block_to_right_offset[CONFLICT_FREE_INDEX(split_indices_block_size_data_partition)];
    }
    if (read_index < num_blocks) {
      const unsigned int conflict_free_threadIdx_x_plus_1 = CONFLICT_FREE_INDEX(threadIdx_x + 1);
      block_to_right_offset_buffer[read_index + 1] = block_to_right_offset[conflict_free_threadIdx_x_plus_1];
    }
    if (read_index_plus_blockDim_x < num_blocks) {
      const unsigned int conflict_free_threadIdx_x_plus_1_plus_blockDim_x = CONFLICT_FREE_INDEX(threadIdx_x + 1 + blockDim_x);
      block_to_right_offset_buffer[read_index_plus_blockDim_x + 1] = block_to_right_offset[conflict_free_threadIdx_x_plus_1_plus_blockDim_x];
    }
    __syncthreads();
  }
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    const int cur_max_leaf_index = (*cuda_cur_num_leaves);
    block_to_left_offset_buffer[0] = 0;
    const unsigned int to_left_total_cnt = block_to_left_offset_buffer[num_blocks];
    block_to_right_offset_buffer[0] = to_left_total_cnt;
    const data_size_t old_leaf_data_end = cuda_leaf_data_end[leaf_index_ref];
    cuda_leaf_data_end[leaf_index_ref] = cuda_leaf_data_start[leaf_index_ref] + static_cast<data_size_t>(to_left_total_cnt);
    cuda_leaf_num_data[leaf_index_ref] = static_cast<data_size_t>(to_left_total_cnt);
    cuda_leaf_data_start[cur_max_leaf_index] = cuda_leaf_data_end[leaf_index_ref];
    cuda_leaf_data_end[cur_max_leaf_index] = old_leaf_data_end;
    cuda_leaf_num_data[cur_max_leaf_index] = block_to_right_offset_buffer[num_blocks] - to_left_total_cnt;
    const int cuda_num_total_bin_ref = *cuda_num_total_bin;
    ++(*cuda_cur_num_leaves);
    if (cuda_leaf_num_data[leaf_index_ref] < cuda_leaf_num_data[cur_max_leaf_index]) {
      *smaller_leaf_cuda_leaf_index_pointer = leaf_index_ref;
      *smaller_leaf_cuda_sum_of_gradients_pointer = best_left_sum_gradients[leaf_index_ref];
      *smaller_leaf_cuda_sum_of_hessians_pointer = best_left_sum_hessians[leaf_index_ref];
      *smaller_leaf_cuda_num_data_in_leaf_pointer = to_left_total_cnt;//best_left_count[leaf_index_ref];
      *smaller_leaf_cuda_gain_pointer = best_left_gain[leaf_index_ref];
      *smaller_leaf_cuda_leaf_value_pointer = best_left_leaf_value[leaf_index_ref];
      *smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer = cuda_data_indices + cuda_leaf_data_start[leaf_index_ref];

      *larger_leaf_cuda_leaf_index_pointer = cur_max_leaf_index;
      *larger_leaf_cuda_sum_of_gradients_pointer = best_right_sum_gradients[leaf_index_ref];
      *larger_leaf_cuda_sum_of_hessians_pointer = best_right_sum_hessians[leaf_index_ref];
      *larger_leaf_cuda_num_data_in_leaf_pointer = cuda_leaf_num_data[cur_max_leaf_index];//best_right_count[leaf_index_ref];
      *larger_leaf_cuda_gain_pointer = best_right_gain[leaf_index_ref];
      *larger_leaf_cuda_leaf_value_pointer = best_right_leaf_value[leaf_index_ref];
      *larger_leaf_cuda_data_indices_in_leaf_pointer_pointer = cuda_data_indices + cuda_leaf_data_start[cur_max_leaf_index];

      hist_t* parent_hist_ptr = cuda_hist_pool[leaf_index_ref];
      cuda_hist_pool[cur_max_leaf_index] = parent_hist_ptr;
      cuda_hist_pool[leaf_index_ref] = cuda_hist + 2 * cur_max_leaf_index * cuda_num_total_bin_ref;
      *smaller_leaf_cuda_hist_pointer_pointer = cuda_hist_pool[leaf_index_ref];
      *larger_leaf_cuda_hist_pointer_pointer = cuda_hist_pool[cur_max_leaf_index];
    } else {
      *larger_leaf_cuda_leaf_index_pointer = leaf_index_ref;
      *larger_leaf_cuda_sum_of_gradients_pointer = best_left_sum_gradients[leaf_index_ref];
      *larger_leaf_cuda_sum_of_hessians_pointer = best_left_sum_hessians[leaf_index_ref];
      *larger_leaf_cuda_num_data_in_leaf_pointer = to_left_total_cnt;//best_left_count[leaf_index_ref];
      *larger_leaf_cuda_gain_pointer = best_left_gain[leaf_index_ref];
      *larger_leaf_cuda_leaf_value_pointer = best_left_leaf_value[leaf_index_ref];
      *larger_leaf_cuda_data_indices_in_leaf_pointer_pointer = cuda_data_indices + cuda_leaf_data_start[leaf_index_ref];

      *smaller_leaf_cuda_leaf_index_pointer = cur_max_leaf_index;
      *smaller_leaf_cuda_sum_of_gradients_pointer = best_right_sum_gradients[leaf_index_ref];
      *smaller_leaf_cuda_sum_of_hessians_pointer = best_right_sum_hessians[leaf_index_ref];
      *smaller_leaf_cuda_num_data_in_leaf_pointer = cuda_leaf_num_data[cur_max_leaf_index];//best_right_count[leaf_index_ref];
      *smaller_leaf_cuda_gain_pointer = best_right_gain[leaf_index_ref];
      *smaller_leaf_cuda_leaf_value_pointer = best_right_leaf_value[leaf_index_ref];
      *smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer = cuda_data_indices + cuda_leaf_data_start[cur_max_leaf_index];

      cuda_hist_pool[cur_max_leaf_index] = cuda_hist + 2 * cur_max_leaf_index * cuda_num_total_bin_ref;
      *smaller_leaf_cuda_hist_pointer_pointer = cuda_hist_pool[cur_max_leaf_index];
      *larger_leaf_cuda_hist_pointer_pointer = cuda_hist_pool[leaf_index_ref];
    }
  }
}

__global__ void SplitInnerKernel(const int* leaf_index, const int* cuda_cur_num_leaves,
  const data_size_t* cuda_leaf_data_start, const data_size_t* cuda_leaf_num_data,
  const data_size_t* cuda_data_indices, const uint8_t* split_to_left_bit_vector,
  const data_size_t* block_to_left_offset_buffer, const data_size_t* block_to_right_offset_buffer,
  data_size_t* out_data_indices_in_leaf, const int split_indices_block_size_data_partition) {
  __shared__ uint8_t thread_split_to_left_bit_vector[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION];
  __shared__ uint32_t thread_to_left_pos[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 1 +
    (SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 2) / NUM_BANKS_DATA_PARTITION];
  __shared__ uint32_t thread_to_right_pos[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 1 +
    (SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 2) / NUM_BANKS_DATA_PARTITION];
  const int leaf_index_ref = *leaf_index;
  const data_size_t leaf_num_data_offset = cuda_leaf_data_start[leaf_index_ref];
  const data_size_t num_data_in_leaf_ref = cuda_leaf_num_data[leaf_index_ref] + cuda_leaf_num_data[(*cuda_cur_num_leaves) - 1];
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int blockDim_x = blockDim.x;
  const unsigned int conflict_free_threadIdx_x_plus_1 = CONFLICT_FREE_INDEX(threadIdx_x + 1);
  const unsigned int global_thread_index = blockIdx.x * blockDim_x * 2 + threadIdx_x;
  const data_size_t* cuda_data_indices_in_leaf = cuda_data_indices + leaf_num_data_offset;
  if (global_thread_index < num_data_in_leaf_ref) {
    const uint8_t bit = split_to_left_bit_vector[global_thread_index];
    thread_split_to_left_bit_vector[threadIdx_x] = bit;
    thread_to_left_pos[conflict_free_threadIdx_x_plus_1] = bit;
    thread_to_right_pos[conflict_free_threadIdx_x_plus_1] = 1 - bit;
  } else {
    thread_split_to_left_bit_vector[threadIdx_x] = 0;
    thread_to_left_pos[conflict_free_threadIdx_x_plus_1] = 0;
    thread_to_right_pos[conflict_free_threadIdx_x_plus_1] = 0;
  }
  const unsigned int conflict_free_threadIdx_x_plus_blockDim_x_plus_1 = CONFLICT_FREE_INDEX(threadIdx_x + blockDim_x + 1);
  const unsigned int global_thread_index_plus_blockDim_x = global_thread_index + blockDim_x;
  if (global_thread_index_plus_blockDim_x < num_data_in_leaf_ref) {
    const uint8_t bit = split_to_left_bit_vector[global_thread_index_plus_blockDim_x];
    thread_split_to_left_bit_vector[threadIdx_x + blockDim_x] = bit;
    thread_to_left_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1] = bit;
    thread_to_right_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1] = 1 - bit;
  } else {
    thread_split_to_left_bit_vector[threadIdx_x + blockDim_x] = 0;
    thread_to_left_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1] = 0;
    thread_to_right_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1] = 0;
  }
  __syncthreads();
  if (threadIdx_x == 0) {
    const uint32_t to_right_block_offset = block_to_right_offset_buffer[blockIdx.x];
    const uint32_t to_left_block_offset = block_to_left_offset_buffer[blockIdx.x];
    thread_to_left_pos[0] = to_left_block_offset;
    thread_to_right_pos[0] = to_right_block_offset;
  }
  __syncthreads();
  PrefixSum(thread_to_left_pos, split_indices_block_size_data_partition);
  PrefixSum(thread_to_right_pos, split_indices_block_size_data_partition);
  __syncthreads();
  if (global_thread_index < num_data_in_leaf_ref) {
    if (thread_split_to_left_bit_vector[threadIdx_x] == 1) {
      out_data_indices_in_leaf[thread_to_left_pos[conflict_free_threadIdx_x_plus_1]] = cuda_data_indices_in_leaf[global_thread_index];
    } else {
      out_data_indices_in_leaf[thread_to_right_pos[conflict_free_threadIdx_x_plus_1]] = cuda_data_indices_in_leaf[global_thread_index];
    }
  }
  if (global_thread_index_plus_blockDim_x < num_data_in_leaf_ref) {
    if (thread_split_to_left_bit_vector[threadIdx_x + blockDim_x] == 1) {
      out_data_indices_in_leaf[thread_to_left_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1]] = cuda_data_indices_in_leaf[global_thread_index_plus_blockDim_x];
    } else {
      out_data_indices_in_leaf[thread_to_right_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1]] = cuda_data_indices_in_leaf[global_thread_index_plus_blockDim_x];
    }
  }
  /*if (thread_to_left_pos[conflict_free_threadIdx_x_plus_1] == 0) {
    printf("thread_to_left_pos[%d] = %d, global_thread_index = %d, thread_split_to_left_bit_vector[%d] = %d\n",
    conflict_free_threadIdx_x_plus_1, thread_to_left_pos[conflict_free_threadIdx_x_plus_1], global_thread_index, threadIdx_x, thread_split_to_left_bit_vector[threadIdx_x]);
  }
  if (thread_to_right_pos[conflict_free_threadIdx_x_plus_1] == 0) {
    printf("thread_to_right_pos[%d] = %d, global_thread_index = %d, thread_split_to_left_bit_vector[%d] = %d\n",
    conflict_free_threadIdx_x_plus_1, thread_to_left_pos[conflict_free_threadIdx_x_plus_1], global_thread_index, threadIdx_x, thread_split_to_left_bit_vector[threadIdx_x]);
  }
  if (thread_to_left_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1] == 0) {
    printf("thread_to_left_pos[%d] = %d, global_thread_index = %d, thread_split_to_left_bit_vector[%d + %ds] = %d\n",
    conflict_free_threadIdx_x_plus_blockDim_x_plus_1, thread_to_left_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1], global_thread_index_plus_blockDim_x, threadIdx_x, blockDim_x, thread_split_to_left_bit_vector[threadIdx_x + blockDim_x]);
  }
  if (thread_to_right_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1] == 0) {
    printf("thread_to_right_pos[%d] = %d, global_thread_index = %d, thread_split_to_left_bit_vector[%d + %d] = %d\n",
    conflict_free_threadIdx_x_plus_blockDim_x_plus_1, thread_to_left_pos[conflict_free_threadIdx_x_plus_blockDim_x_plus_1], global_thread_index_plus_blockDim_x, threadIdx_x, blockDim_x, thread_split_to_left_bit_vector[threadIdx_x + blockDim_x]);
  }*/
}

/*__global__ void SplitInnerKernel(const int* leaf_index, const int* cuda_cur_num_leaves,
  const data_size_t* cuda_leaf_data_start, const data_size_t* cuda_leaf_num_data,
  const data_size_t* cuda_data_indices, const uint8_t* split_to_left_bit_vector,
  const data_size_t* block_to_left_offset_buffer, const data_size_t* block_to_right_offset_buffer,
  data_size_t* out_data_indices_in_leaf) {
  __shared__ uint8_t thread_split_to_left_bit_vector[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION];
  __shared__ uint32_t thread_to_left_pos[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION];
  __shared__ uint32_t thread_to_right_pos[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION];
  const int leaf_index_ref = *leaf_index;
  const data_size_t leaf_num_data_offset = cuda_leaf_data_start[leaf_index_ref];
  const data_size_t num_data_in_leaf_ref = cuda_leaf_num_data[leaf_index_ref] + cuda_leaf_num_data[(*cuda_cur_num_leaves) - 1];
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int global_thread_index = blockIdx.x * blockDim.x + threadIdx_x;
  const data_size_t* cuda_data_indices_in_leaf = cuda_data_indices + leaf_num_data_offset;
  if (global_thread_index < num_data_in_leaf_ref) {
    thread_split_to_left_bit_vector[threadIdx_x] = split_to_left_bit_vector[global_thread_index];
  } else {
    thread_split_to_left_bit_vector[threadIdx_x] = 0;
  }
  __syncthreads();
  if (threadIdx_x == 0) {
    const uint32_t to_right_block_offset = block_to_right_offset_buffer[blockIdx.x];
    const uint32_t to_left_block_offset = block_to_left_offset_buffer[blockIdx.x];
    thread_to_left_pos[0] = to_left_block_offset;
    thread_to_right_pos[0] = to_right_block_offset;
    for (unsigned int i = 0; i < blockDim.x - 1; ++i) {
      const unsigned int tmp_global_thread_index = blockIdx.x * blockDim.x + i;
      if (tmp_global_thread_index < num_data_in_leaf_ref) {
        if (thread_split_to_left_bit_vector[i] == 0) {
          thread_to_right_pos[i + 1] = thread_to_right_pos[i] + 1;
          thread_to_left_pos[i + 1] = thread_to_left_pos[i];
        } else {
          thread_to_left_pos[i + 1] = thread_to_left_pos[i] + 1;
          thread_to_right_pos[i + 1] = thread_to_right_pos[i];
        }
      } else {
        thread_to_left_pos[i + 1] = thread_to_left_pos[i];
        thread_to_right_pos[i + 1] = thread_to_right_pos[i];
      }
    }
  }
  __syncthreads();
  if (global_thread_index < num_data_in_leaf_ref) {
    if (thread_split_to_left_bit_vector[threadIdx_x] == 1) {
      out_data_indices_in_leaf[thread_to_left_pos[threadIdx_x]] = cuda_data_indices_in_leaf[global_thread_index];
    } else {
      out_data_indices_in_leaf[thread_to_right_pos[threadIdx_x]] = cuda_data_indices_in_leaf[global_thread_index];
    }
  }
}*/

__global__ void CopyDataIndicesKernel(const int* leaf_index,
  const int* cuda_cur_num_leaves,
  const data_size_t* cuda_leaf_data_start,
  const data_size_t* cuda_leaf_num_data,
  const data_size_t* out_data_indices_in_leaf,
  data_size_t* cuda_data_indices) {
  const int leaf_index_ref = *leaf_index;
  const data_size_t leaf_num_data_offset = cuda_leaf_data_start[leaf_index_ref];
  const data_size_t num_data_in_leaf_ref = cuda_leaf_num_data[leaf_index_ref] + cuda_leaf_num_data[(*cuda_cur_num_leaves) - 1];
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int global_thread_index = blockIdx.x * blockDim.x + threadIdx_x;
  data_size_t* cuda_data_indices_in_leaf = cuda_data_indices + leaf_num_data_offset;
  if (global_thread_index < num_data_in_leaf_ref) {
    cuda_data_indices_in_leaf[global_thread_index] = out_data_indices_in_leaf[global_thread_index];
  }
}

void CUDADataPartition::LaunchSplitInnerKernel(const int* leaf_index, const data_size_t num_data_in_leaf,
  const double* best_left_sum_gradients, const double* best_left_sum_hessians, const data_size_t* best_left_count,
  const double* best_left_gain, const double* best_left_leaf_value,
  const double* best_right_sum_gradients, const double* best_right_sum_hessians, const data_size_t* best_right_count,
  const double* best_right_gain, const double* best_right_leaf_value,
  // for leaf splits information update
  int* smaller_leaf_cuda_leaf_index_pointer, double* smaller_leaf_cuda_sum_of_gradients_pointer,
  double* smaller_leaf_cuda_sum_of_hessians_pointer, data_size_t* smaller_leaf_cuda_num_data_in_leaf_pointer,
  double* smaller_leaf_cuda_gain_pointer, double* smaller_leaf_cuda_leaf_value_pointer,
  const data_size_t** smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer,
  hist_t** smaller_leaf_cuda_hist_pointer_pointer,
  int* larger_leaf_cuda_leaf_index_pointer, double* larger_leaf_cuda_sum_of_gradients_pointer,
  double* larger_leaf_cuda_sum_of_hessians_pointer, data_size_t* larger_leaf_cuda_num_data_in_leaf_pointer,
  double* larger_leaf_cuda_gain_pointer, double* larger_leaf_cuda_leaf_value_pointer,
  const data_size_t** larger_leaf_cuda_data_indices_in_leaf_pointer_pointer,
  hist_t** larger_leaf_cuda_hist_pointer_pointer, const int cpu_leaf_index) {
  //Log::Warning("num_data_in_leaf = %d", num_data_in_leaf);
  const int num_blocks = std::max(80, (num_data_in_leaf + SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION - 1) / SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION);
  int split_indices_block_size_data_partition = (num_data_in_leaf + num_blocks - 1) / num_blocks - 1;
  int split_indices_block_size_data_partition_aligned = 1;
  while (split_indices_block_size_data_partition > 0) {
    split_indices_block_size_data_partition_aligned <<= 1;
    split_indices_block_size_data_partition >>= 1;
  }
  //Log::Warning("num_blocks = %d, split_indices_block_size_data_partition_aligned = %d", num_blocks, split_indices_block_size_data_partition_aligned);
  auto start = std::chrono::steady_clock::now();
  const int num_blocks_final = (num_data_in_leaf + split_indices_block_size_data_partition_aligned - 1) / split_indices_block_size_data_partition_aligned;
  //Log::Warning("num_blocks_final = %d", num_blocks_final);
  PrepareOffsetKernel<<<num_blocks_final, split_indices_block_size_data_partition_aligned / 2>>>(
    leaf_index, cuda_leaf_num_data_, cuda_data_to_left_,
    cuda_block_data_to_left_offset_, cuda_block_data_to_right_offset_, split_indices_block_size_data_partition_aligned);
  SynchronizeCUDADevice();
  auto end = std::chrono::steady_clock::now();
  double duration = (static_cast<std::chrono::duration<double>>(end - start)).count();
  //Log::Warning("CUDADataPartition::PrepareOffsetKernel time %f", duration);
  start = std::chrono::steady_clock::now();
  AggregateBlockOffsetKernel<<<1, split_indices_block_size_data_partition_aligned / 2>>>(leaf_index, cuda_block_data_to_left_offset_,
    cuda_block_data_to_right_offset_, cuda_leaf_data_start_, cuda_leaf_data_end_,
    cuda_leaf_num_data_, cuda_data_indices_,
    cuda_cur_num_leaves_,

    best_left_sum_gradients, best_left_sum_hessians, best_left_count,
    best_left_gain, best_left_leaf_value,
    best_right_sum_gradients, best_right_sum_hessians, best_right_count,
    best_right_gain, best_right_leaf_value,

    smaller_leaf_cuda_leaf_index_pointer, smaller_leaf_cuda_sum_of_gradients_pointer,
    smaller_leaf_cuda_sum_of_hessians_pointer, smaller_leaf_cuda_num_data_in_leaf_pointer,
    smaller_leaf_cuda_gain_pointer, smaller_leaf_cuda_leaf_value_pointer,
    smaller_leaf_cuda_data_indices_in_leaf_pointer_pointer,
    smaller_leaf_cuda_hist_pointer_pointer,
    larger_leaf_cuda_leaf_index_pointer, larger_leaf_cuda_sum_of_gradients_pointer,
    larger_leaf_cuda_sum_of_hessians_pointer, larger_leaf_cuda_num_data_in_leaf_pointer,
    larger_leaf_cuda_gain_pointer, larger_leaf_cuda_leaf_value_pointer,
    larger_leaf_cuda_data_indices_in_leaf_pointer_pointer,
    larger_leaf_cuda_hist_pointer_pointer,
    cuda_num_total_bin_,
    cuda_hist_,
    cuda_hist_pool_, split_indices_block_size_data_partition_aligned);
  const auto copy_start = std::chrono::steady_clock::now();
  CopyFromCUDADeviceToHost<data_size_t>(num_data_in_leaf_.data(), cuda_leaf_num_data_, num_leaves_);
  SynchronizeCUDADevice();
  const auto copy_end = std::chrono::steady_clock::now();
  const auto copy_duration = (static_cast<std::chrono::duration<double>>(copy_end - copy_start)).count();
  //Log::Warning("CUDADataPartition::CopyFromCUDADeviceToHost time %f", copy_duration);
  end = std::chrono::steady_clock::now();
  duration = (static_cast<std::chrono::duration<double>>(end - start)).count();
  //Log::Warning("CUDADataPartition::AggregateBlockOffsetKernel time %f", duration);
  start = std::chrono::steady_clock::now();
  SplitInnerKernel<<<num_blocks_final, split_indices_block_size_data_partition_aligned / 2>>>(
    leaf_index, cuda_cur_num_leaves_, cuda_leaf_data_start_, cuda_leaf_num_data_, cuda_data_indices_, cuda_data_to_left_,
    cuda_block_data_to_left_offset_, cuda_block_data_to_right_offset_,
    cuda_out_data_indices_in_leaf_, split_indices_block_size_data_partition_aligned);
  SynchronizeCUDADevice();
  end = std::chrono::steady_clock::now();
  duration = (static_cast<std::chrono::duration<double>>(end - start)).count();
  //Log::Warning("CUDADataPartition::SplitInnerKernel time %f", duration);
  start = std::chrono::steady_clock::now();
  CopyDataIndicesKernel<<<num_blocks_final, split_indices_block_size_data_partition_aligned>>>(
    leaf_index, cuda_cur_num_leaves_, cuda_leaf_data_start_, cuda_leaf_num_data_, cuda_out_data_indices_in_leaf_, cuda_data_indices_);
  SynchronizeCUDADevice();
  end = std::chrono::steady_clock::now();
  duration = (static_cast<std::chrono::duration<double>>(end - start)).count();
  //Log::Warning("CUDADataPartition::CopyDataIndicesKernel time %f", duration);
}

__global__ void PrefixSumKernel(uint32_t* cuda_elements) {
  __shared__ uint32_t elements[SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION + 1];
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int global_read_index = blockIdx.x * blockDim.x * 2 + threadIdx_x;
  elements[threadIdx_x] = cuda_elements[global_read_index];
  elements[threadIdx_x + blockDim.x] = cuda_elements[global_read_index + blockDim.x];
  __syncthreads();
  PrefixSum(elements, SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION);
  __syncthreads();
  cuda_elements[global_read_index] = elements[threadIdx_x];
  cuda_elements[global_read_index + blockDim.x] = elements[threadIdx_x + blockDim.x];
}

void CUDADataPartition::LaunchPrefixSumKernel(uint32_t* cuda_elements) {
  PrefixSumKernel<<<1, SPLIT_INDICES_BLOCK_SIZE_DATA_PARTITION / 2>>>(cuda_elements);
  SynchronizeCUDADevice();
}

__global__ void AddPredictionToScoreKernel(const double* data_partition_leaf_output,
  const data_size_t* num_data_in_leaf, const data_size_t* data_indices_in_leaf,
  const data_size_t* leaf_data_start, const double learning_rate, double* output_score) {
  const unsigned int threadIdx_x = threadIdx.x;
  const unsigned int blockIdx_x = blockIdx.x;
  const unsigned int blockDim_x = blockDim.x;
  const data_size_t num_data = num_data_in_leaf[blockIdx_x];
  const data_size_t* data_indices = data_indices_in_leaf + leaf_data_start[blockIdx_x];
  const double leaf_prediction_value = data_partition_leaf_output[blockIdx_x] * learning_rate;
  /*if (threadIdx_x == 0) {
    printf("leaf index = %d, leaf_prediction_value = %f\n", blockIdx_x, leaf_prediction_value);
  }*/
  for (unsigned int offset = 0; offset < static_cast<unsigned int>(num_data); offset += blockDim_x) {
    const data_size_t inner_data_index = static_cast<data_size_t>(offset + threadIdx_x);
    if (inner_data_index < num_data) {
      const data_size_t data_index = data_indices[inner_data_index];
      output_score[data_index] = leaf_prediction_value;
    }
  }
}

void CUDADataPartition::LaunchAddPredictionToScoreKernel(const double learning_rate) {
  AddPredictionToScoreKernel<<<cur_num_leaves_, 1024>>>(data_partition_leaf_output_,
    cuda_leaf_num_data_, cuda_data_indices_, cuda_leaf_data_start_, learning_rate, train_data_score_tmp_);
  SynchronizeCUDADevice();
  CopyFromCUDADeviceToHost<double>(cpu_train_data_score_tmp_.data(), train_data_score_tmp_, static_cast<size_t>(num_data_));
  SynchronizeCUDADevice();
}

}  // namespace LightGBM

#endif  // USE_CUDA
