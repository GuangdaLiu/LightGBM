
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 */

#ifdef USE_CUDA

#include "cuda_best_split_finder.hpp"

namespace LightGBM {

#define K_MIN_SCORE (-1000000.0)

#define K_EPSILON (1e-15f)

#define CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(n) \
  ((n) + ((n) >> LOG_NUM_BANKS_DATA_PARTITION_BEST_SPLIT_FINDER)) \

__device__ void PrefixSumHist(hist_t* elements, unsigned int n) {
  unsigned int offset = 1;
  unsigned int threadIdx_x = threadIdx.x;
  const unsigned int conflict_free_n_minus_1 = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(n - 1);
  const hist_t last_element = elements[conflict_free_n_minus_1];
  __syncthreads();
  for (int d = (n >> 1); d > 0; d >>= 1) {
    if (threadIdx_x < d) {
      const unsigned int src_pos = offset * (2 * threadIdx_x + 1) - 1;
      const unsigned int dst_pos = offset * (2 * threadIdx_x + 2) - 1;
      elements[CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(dst_pos)] += elements[CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(src_pos)];
    }
    offset <<= 1;
    __syncthreads();
  }
  if (threadIdx_x == 0) {
    elements[conflict_free_n_minus_1] = 0; 
  }
  __syncthreads();
  for (int d = 1; d < n; d <<= 1) {
    offset >>= 1;
    if (threadIdx_x < d) {
      const unsigned int dst_pos = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(offset * (2 * threadIdx_x + 2) - 1);
      const unsigned int src_pos = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(offset * (2 * threadIdx_x + 1) - 1);
      const hist_t src_val = elements[src_pos];
      elements[src_pos] = elements[dst_pos];
      elements[dst_pos] += src_val;
    }
    __syncthreads();
  }
  if (threadIdx_x == 0) {
    elements[CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(n)] = elements[conflict_free_n_minus_1] + last_element;
  }
  __syncthreads();
}

__device__ void PrefixSumHistCnt(data_size_t* elements, unsigned int n) {
  unsigned int offset = 1;
  unsigned int threadIdx_x = threadIdx.x;
  const unsigned int conflict_free_n_minus_1 = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(n - 1);
  const data_size_t last_element = elements[conflict_free_n_minus_1];
  __syncthreads();
  for (int d = (n >> 1); d > 0; d >>= 1) {
    if (threadIdx_x < d) {
      const unsigned int src_pos = offset * (2 * threadIdx_x + 1) - 1;
      const unsigned int dst_pos = offset * (2 * threadIdx_x + 2) - 1;
      elements[CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(dst_pos)] += elements[CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(src_pos)];
    }
    offset <<= 1;
    __syncthreads();
  }
  if (threadIdx_x == 0) {
    elements[conflict_free_n_minus_1] = 0; 
  }
  __syncthreads();
  for (int d = 1; d < n; d <<= 1) {
    offset >>= 1;
    if (threadIdx_x < d) {
      const unsigned int dst_pos = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(offset * (2 * threadIdx_x + 2) - 1);
      const unsigned int src_pos = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(offset * (2 * threadIdx_x + 1) - 1);
      const data_size_t src_val = elements[src_pos];
      elements[src_pos] = elements[dst_pos];
      elements[dst_pos] += src_val;
    }
    __syncthreads();
  }
  if (threadIdx_x == 0) {
    elements[CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(n)] = elements[conflict_free_n_minus_1] + last_element;
  }
}

__device__ void ReduceBestGain(double* gain, hist_t* sum_gradients,
  hist_t* sum_hessians, /*data_size_t* num_data,*/ uint8_t* found,
  uint32_t* threshold_value) {
  const unsigned int tid = threadIdx.x;
  const unsigned int conflict_free_tid_plus_1 = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(tid + 1);
  for (unsigned int s = 1; s < MAX_NUM_BIN_IN_FEATURE; s *= 2) {
    if (tid % (2 * s) == 0 && (tid + s) < MAX_NUM_BIN_IN_FEATURE) {
      const uint32_t tid_s = tid + s;
      const uint32_t conflict_free_tid_s_plus_1 = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(tid_s + 1);
      if ((found[tid_s] && !found[tid]) || (found[tid_s] && found[tid] && gain[tid_s] > gain[tid])) {
        gain[tid] = gain[tid_s];
        sum_gradients[conflict_free_tid_plus_1] = sum_gradients[conflict_free_tid_s_plus_1];
        sum_hessians[conflict_free_tid_plus_1] = sum_hessians[conflict_free_tid_s_plus_1];
        found[tid] = found[tid_s];
        threshold_value[tid] = threshold_value[tid_s];
      }
    }
    __syncthreads();
  }
}

__device__ void ReduceBestGainForLeaves(double* gain, int* leaves, int cuda_cur_num_leaves) {
  const unsigned int tid = threadIdx.x;
  for (unsigned int s = 1; s < cuda_cur_num_leaves; s *= 2) {
    if (tid % (2 * s) == 0 && (tid + s) < cuda_cur_num_leaves) {
      const uint32_t tid_s = tid + s;
      if ((leaves[tid] == -1 && leaves[tid_s] != -1) || (leaves[tid] != -1 && leaves[tid_s] != -1 && gain[tid_s] > gain[tid])) {
        gain[tid] = gain[tid_s];
        leaves[tid] = leaves[tid_s];
      }
    }
    __syncthreads();
  }
}

__device__ double ThresholdL1(double s, double l1) {
  const double reg_s = fmax(0.0, fabs(s) - l1);
  if (s >= 0.0f) {
    return reg_s;
  } else {
    return -reg_s;
  }
}

__device__ double CalculateSplittedLeafOutput(double sum_gradients,
                                          double sum_hessians, double l1, const bool use_l1,
                                          double l2) {
  double ret;
  if (use_l1) {
    ret = -ThresholdL1(sum_gradients, l1) / (sum_hessians + l2);
  } else {
    ret = -sum_gradients / (sum_hessians + l2);
  }
  return ret;
}

__device__ double GetLeafGainGivenOutput(double sum_gradients,
                                      double sum_hessians, double l1, const bool use_l1,
                                      double l2, double output) {
  if (use_l1) {
    const double sg_l1 = ThresholdL1(sum_gradients, l1);
    return -(2.0 * sg_l1 * output + (sum_hessians + l2) * output * output);
  } else {
    return -(2.0 * sum_gradients * output +
              (sum_hessians + l2) * output * output);
  }
}

__device__ double GetLeafGain(double sum_gradients, double sum_hessians,
                          double l1, const bool use_l1, double l2) {
  if (use_l1) {
    const double sg_l1 = ThresholdL1(sum_gradients, l1);
    return (sg_l1 * sg_l1) / (sum_hessians + l2);
  } else {
    return (sum_gradients * sum_gradients) / (sum_hessians + l2);
  }
}

__device__ double GetSplitGains(double sum_left_gradients,
                            double sum_left_hessians,
                            double sum_right_gradients,
                            double sum_right_hessians,
                            double l1, const bool use_l1, double l2) {
  return GetLeafGain(sum_left_gradients,
                     sum_left_hessians,
                     l1, use_l1, l2) +
         GetLeafGain(sum_right_gradients,
                     sum_right_hessians,
                     l1, use_l1, l2);
}

__device__ void FindBestSplitsForLeafKernelInner(
  // input feature information
  const hist_t* feature_hist_ptr,
  const uint32_t feature_num_bin,
  const uint8_t feature_mfb_offset,
  const uint32_t feature_default_bin,
  const uint8_t feature_missing_type,
  const int inner_feature_index,
  // input config parameter values
  const double lambda_l1,
  const double lambda_l2,
  const data_size_t min_data_in_leaf,
  const double min_sum_hessian_in_leaf,
  const double min_gain_to_split,
  // input parent node information
  const double parent_gain,
  const double sum_gradients,
  const double sum_hessians,
  const data_size_t num_data,
  // input task information
  const bool reverse,
  const bool skip_default_bin,
  const bool na_as_missing,
  const uint8_t assume_out_default_left,
  // output parameters
  CUDASplitInfo* cuda_best_split_info) {

  const double cnt_factor = num_data / sum_hessians;
  const bool use_l1 = lambda_l1 > 0.0f;
  const double min_gain_shift = parent_gain + min_gain_to_split;

  cuda_best_split_info->is_valid = false;

  __shared__ hist_t local_grad_hist[MAX_NUM_BIN_IN_FEATURE + 1 + (MAX_NUM_BIN_IN_FEATURE + 1) / LOG_NUM_BANKS_DATA_PARTITION_BEST_SPLIT_FINDER];
  __shared__ hist_t local_hess_hist[MAX_NUM_BIN_IN_FEATURE + 1 + (MAX_NUM_BIN_IN_FEATURE + 1) / LOG_NUM_BANKS_DATA_PARTITION_BEST_SPLIT_FINDER];
  __shared__ double local_gain[MAX_NUM_BIN_IN_FEATURE];
  __shared__ uint8_t threshold_found[MAX_NUM_BIN_IN_FEATURE];
  __shared__ uint32_t threshold_value[MAX_NUM_BIN_IN_FEATURE];

  const unsigned int threadIdx_x = threadIdx.x;
  const bool skip_sum = (skip_default_bin && (threadIdx_x + feature_mfb_offset) == static_cast<int>(feature_default_bin));
  const uint32_t feature_num_bin_minus_offset = feature_num_bin - feature_mfb_offset;
  const bool skip_split = (skip_default_bin && (feature_num_bin_minus_offset - 1 - threadIdx_x + feature_mfb_offset == static_cast<int>(feature_default_bin)));
  const unsigned int bin_offset = threadIdx_x << 1;
  const unsigned int conflict_free_threadIdx_x = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(threadIdx_x);
  if (!reverse) {
    if (threadIdx_x < feature_num_bin_minus_offset && !skip_sum) {
      local_grad_hist[conflict_free_threadIdx_x] = feature_hist_ptr[bin_offset];
      const hist_t hess = feature_hist_ptr[bin_offset + 1];
      local_hess_hist[conflict_free_threadIdx_x] = hess;
    } else {
      local_grad_hist[conflict_free_threadIdx_x] = 0.0f;
      local_hess_hist[conflict_free_threadIdx_x] = 0.0f;
    }
  } else {
    if (threadIdx_x < feature_num_bin_minus_offset) {
      const unsigned int write_index = feature_num_bin_minus_offset - 1 - threadIdx_x;
      const unsigned int conflict_free_write_index = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(write_index);
      if (!skip_sum) {
        local_grad_hist[conflict_free_write_index] = feature_hist_ptr[bin_offset];
        const hist_t hess = feature_hist_ptr[bin_offset + 1];
        local_hess_hist[conflict_free_write_index] = hess;
      } else {
        local_grad_hist[conflict_free_write_index] = 0.0f;
        local_hess_hist[conflict_free_write_index] = 0.0f;
      }
    } else {
      local_grad_hist[conflict_free_threadIdx_x] = 0.0f;
      local_hess_hist[conflict_free_threadIdx_x] = 0.0f;
    }
  }
  __syncthreads();
  if (threadIdx_x == 0) {
    local_hess_hist[conflict_free_threadIdx_x] += K_EPSILON;
  }
  local_gain[threadIdx_x] = K_MIN_SCORE;
  __syncthreads();
  PrefixSumHist(local_grad_hist, MAX_NUM_BIN_IN_FEATURE);
  PrefixSumHist(local_hess_hist, MAX_NUM_BIN_IN_FEATURE);
  __syncthreads();
  const unsigned int conflict_free_threadIdx_x_plus_1 = CONFLICT_FREE_INDEX_BEST_SPLIT_FINDER(threadIdx_x + 1);
  if (reverse) {
    if (threadIdx_x >= static_cast<unsigned int>(na_as_missing) && threadIdx_x <= feature_num_bin - 2 && !skip_split) {
      const double sum_right_gradient = local_grad_hist[conflict_free_threadIdx_x_plus_1];
      const double sum_right_hessian = local_hess_hist[conflict_free_threadIdx_x_plus_1];
      const data_size_t right_count = static_cast<data_size_t>(__double2int_rn(sum_right_hessian * cnt_factor));
      const double sum_left_gradient = sum_gradients - sum_right_gradient;
      const double sum_left_hessian = sum_hessians - sum_right_hessian;
      const data_size_t left_count = num_data - right_count;
      if (sum_left_hessian >= min_sum_hessian_in_leaf && left_count >= min_data_in_leaf &&
        sum_right_hessian >= min_sum_hessian_in_leaf && right_count >= min_data_in_leaf) {
        double current_gain = GetSplitGains(
          sum_left_gradient, sum_left_hessian, sum_right_gradient,
          sum_right_hessian, lambda_l1, use_l1,
          lambda_l2);
        // gain with split is worse than without split
        if (current_gain <= min_gain_shift) {
          threshold_found[threadIdx_x] = 0;
        } else {
          local_gain[threadIdx_x] = current_gain - min_gain_shift;
          threshold_value[threadIdx_x] = static_cast<uint32_t>(feature_num_bin - 2 - threadIdx_x);
          threshold_found[threadIdx_x] = 1;
        }
      } else {
        threshold_found[threadIdx_x] = 0;
      }
    } else {
      threshold_found[threadIdx_x] = 0;
    }
  } else {
    if (threadIdx_x <= feature_num_bin_minus_offset - 2 /* TODO(shiyu1994): skip default */) {
      const double sum_left_gradient = local_grad_hist[conflict_free_threadIdx_x_plus_1];
      const double sum_left_hessian = local_hess_hist[conflict_free_threadIdx_x_plus_1];
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian;
      const data_size_t right_count = num_data - left_count;
      if (sum_left_hessian >= min_sum_hessian_in_leaf && left_count >= min_data_in_leaf &&
        sum_right_hessian >= min_sum_hessian_in_leaf && right_count >= min_data_in_leaf) {
        double current_gain = GetSplitGains(
          sum_left_gradient, sum_left_hessian, sum_right_gradient,
          sum_right_hessian, lambda_l1, use_l1,
          lambda_l2);
        // gain with split is worse than without split
        if (current_gain <= min_gain_shift) {
          threshold_found[threadIdx_x] = 0;
        } else {
          local_gain[threadIdx_x] = current_gain - min_gain_shift;
          threshold_value[threadIdx_x] = static_cast<uint32_t>(threadIdx_x + feature_mfb_offset);
          threshold_found[threadIdx_x] = 1;
        }
      } else {
        threshold_found[threadIdx_x] = 0;
      }
    } else {
      threshold_found[threadIdx_x] = 0;
    }
  }
  __syncthreads();
  ReduceBestGain(local_gain, local_grad_hist, local_hess_hist, threshold_found, threshold_value);
  const uint8_t found = threshold_found[0];
  if (found && threadIdx_x == 0) {
    cuda_best_split_info->is_valid = true;
    cuda_best_split_info->threshold = threshold_value[0];
    cuda_best_split_info->gain = local_gain[0];
    cuda_best_split_info->default_left = assume_out_default_left;
    if (reverse) {
      const double sum_right_gradient = local_grad_hist[1];
      const double sum_right_hessian = local_hess_hist[1] - K_EPSILON;
      const data_size_t right_count = static_cast<data_size_t>(__double2int_rn(sum_right_hessian * cnt_factor));
      const double sum_left_gradient = sum_gradients - sum_right_gradient;
      const double sum_left_hessian = sum_hessians - sum_right_hessian - K_EPSILON;
      const data_size_t left_count = num_data - right_count;
      const double left_output = CalculateSplittedLeafOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2);
      const double right_output = CalculateSplittedLeafOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2);
      cuda_best_split_info->left_sum_gradients = sum_left_gradient;
      cuda_best_split_info->left_sum_hessians = sum_left_hessian;
      cuda_best_split_info->left_count = left_count;
      cuda_best_split_info->right_sum_gradients = sum_right_gradient;
      cuda_best_split_info->right_sum_hessians = sum_right_hessian;
      cuda_best_split_info->right_count = right_count;
      cuda_best_split_info->left_value = left_output;
      cuda_best_split_info->left_gain = GetLeafGainGivenOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2, left_output);
      cuda_best_split_info->right_value = right_output;
      cuda_best_split_info->right_gain = GetLeafGainGivenOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2, right_output);
    } else {
      const double sum_left_gradient = local_grad_hist[1];
      const double sum_left_hessian = local_hess_hist[1] - K_EPSILON;
      const data_size_t left_count = static_cast<data_size_t>(__double2int_rn(sum_left_hessian * cnt_factor));
      const double sum_right_gradient = sum_gradients - sum_left_gradient;
      const double sum_right_hessian = sum_hessians - sum_left_hessian - K_EPSILON;
      const data_size_t right_count = num_data - left_count;
      const double left_output = CalculateSplittedLeafOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2);
      const double right_output = CalculateSplittedLeafOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2);
      cuda_best_split_info->left_sum_gradients = sum_left_gradient;
      cuda_best_split_info->left_sum_hessians = sum_left_hessian;
      cuda_best_split_info->left_count = left_count;
      cuda_best_split_info->right_sum_gradients = sum_right_gradient;
      cuda_best_split_info->right_sum_hessians = sum_right_hessian;
      cuda_best_split_info->right_count = right_count;
      cuda_best_split_info->left_value = left_output;
      cuda_best_split_info->left_gain = GetLeafGainGivenOutput(sum_left_gradient,
        sum_left_hessian, lambda_l1, use_l1, lambda_l2, left_output);
      cuda_best_split_info->right_value = right_output;
      cuda_best_split_info->right_gain = GetLeafGainGivenOutput(sum_right_gradient,
        sum_right_hessian, lambda_l1, use_l1, lambda_l2, right_output);
    }
  }
}

__global__ void FindBestSplitsForLeafKernel(
  // input feature information
  const uint32_t* feature_hist_offsets,
  const uint8_t* feature_mfb_offsets,
  const uint32_t* feature_default_bins, 
  const uint8_t* feature_missing_types,
  const uint32_t* feature_num_bins,
  // input task information
  const bool larger_only,
  const int num_tasks,
  const int* task_feature_index,
  const uint8_t* task_reverse,
  const uint8_t* task_skip_default_bin,
  const uint8_t* task_na_as_missing,
  const uint8_t* task_out_default_left,
  // input leaf information
  const int smaller_leaf_index,
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const int larger_leaf_index,
  const CUDALeafSplitsStruct* larger_leaf_splits,
  // input config parameter values
  const data_size_t min_data_in_leaf,
  const double min_sum_hessian_in_leaf,
  const double min_gain_to_split,
  const double lambda_l1,
  const double lambda_l2,
  // output
  CUDASplitInfo* cuda_best_split_info) {

  const unsigned int task_index = blockIdx.x % num_tasks;
  const bool is_larger = static_cast<bool>(blockIdx.x >= num_tasks || larger_only);
  const int inner_feature_index = task_feature_index[task_index];
  const bool reverse = static_cast<bool>(task_reverse[task_index]);
  const bool skip_default_bin = static_cast<bool>(task_skip_default_bin[task_index]);
  const bool na_as_missing = static_cast<bool>(task_na_as_missing[task_index]);
  const bool assume_out_default_left = task_out_default_left[task_index];
  const double parent_gain = is_larger ? larger_leaf_splits->gain : smaller_leaf_splits->gain;
  const double sum_gradients = is_larger ? larger_leaf_splits->sum_of_gradients : smaller_leaf_splits->sum_of_gradients;
  const double sum_hessians = (is_larger ? larger_leaf_splits->sum_of_hessians : smaller_leaf_splits->sum_of_hessians) + 2 * K_EPSILON;
  const double num_data = is_larger ? larger_leaf_splits->num_data_in_leaf : smaller_leaf_splits->num_data_in_leaf;
  const unsigned int output_offset = is_larger ? (task_index + num_tasks) : task_index;
  CUDASplitInfo* out = cuda_best_split_info + output_offset;
  const hist_t* hist_ptr = (is_larger ? larger_leaf_splits->hist_in_leaf : smaller_leaf_splits->hist_in_leaf) + feature_hist_offsets[inner_feature_index] * 2;
  FindBestSplitsForLeafKernelInner(
    // input feature information
    hist_ptr,
    feature_num_bins[inner_feature_index],
    feature_mfb_offsets[inner_feature_index],
    feature_default_bins[inner_feature_index],
    feature_missing_types[inner_feature_index],
    inner_feature_index,
    // input config parameter values
    lambda_l1,
    lambda_l2,
    min_data_in_leaf,
    min_sum_hessian_in_leaf,
    min_gain_to_split,
    // input parent node information
    parent_gain,
    sum_gradients,
    sum_hessians,
    num_data,
    // input task information
    reverse,
    skip_default_bin,
    na_as_missing,
    assume_out_default_left,
    // output parameters
    out);
}

void CUDABestSplitFinder::LaunchFindBestSplitsForLeafKernel(
  const CUDALeafSplitsStruct* smaller_leaf_splits,
  const CUDALeafSplitsStruct* larger_leaf_splits,
  const int smaller_leaf_index,
  const int larger_leaf_index,
  const bool is_smaller_leaf_valid,
  const bool is_larger_leaf_valid) {
  if (!is_smaller_leaf_valid && !is_larger_leaf_valid) {
    return;
  }
  bool larger_only = false;
  if (!is_smaller_leaf_valid) {
    larger_only = true;
  }
  if (!larger_only) {
    FindBestSplitsForLeafKernel<<<num_tasks_, MAX_NUM_BIN_IN_FEATURE, 0, cuda_streams_[0]>>>(
      // input feature information
      cuda_feature_hist_offsets_,
      cuda_feature_mfb_offsets_,
      cuda_feature_default_bins_,
      cuda_feature_missing_type_,
      cuda_feature_num_bins_,
      // input task information
      larger_only,
      num_tasks_,
      cuda_task_feature_index_,
      cuda_task_reverse_,
      cuda_task_skip_default_bin_,
      cuda_task_na_as_missing_,
      cuda_task_out_default_left_,
      // input leaf information
      smaller_leaf_index,
      smaller_leaf_splits,
      larger_leaf_index,
      larger_leaf_splits,
      // configuration parameter values
      min_data_in_leaf_,
      min_sum_hessian_in_leaf_,
      min_gain_to_split_,
      lambda_l1_,
      lambda_l2_,
      // output parameters
      cuda_best_split_info_);
  }
  SynchronizeCUDADeviceOuter(__FILE__, __LINE__);
  if (larger_leaf_index >= 0) {
    FindBestSplitsForLeafKernel<<<num_tasks_, MAX_NUM_BIN_IN_FEATURE, 0, cuda_streams_[1]>>>(
      // input feature information
      cuda_feature_hist_offsets_,
      cuda_feature_mfb_offsets_,
      cuda_feature_default_bins_,
      cuda_feature_missing_type_,
      cuda_feature_num_bins_,
      // input task information
      true,
      num_tasks_,
      cuda_task_feature_index_,
      cuda_task_reverse_,
      cuda_task_skip_default_bin_,
      cuda_task_na_as_missing_,
      cuda_task_out_default_left_,
      // input leaf information
      smaller_leaf_index,
      smaller_leaf_splits,
      larger_leaf_index,
      larger_leaf_splits,
      // configuration parameter values
      min_data_in_leaf_,
      min_sum_hessian_in_leaf_,
      min_gain_to_split_,
      lambda_l1_,
      lambda_l2_,
      // output parameters
      cuda_best_split_info_);
  }
}

__device__ void ReduceBestSplit(bool* found, double* gain, uint32_t* shared_read_index,
  uint32_t num_features_aligned) {
  const uint32_t threadIdx_x = threadIdx.x;
  for (unsigned int s = 1; s < num_features_aligned; s <<= 1) {
    if (threadIdx_x % (2 * s) == 0 && (threadIdx_x + s) < num_features_aligned) {
      const uint32_t pos_to_compare = threadIdx_x + s;
      if ((!found[threadIdx_x] && found[pos_to_compare]) ||
        (found[threadIdx_x] && found[pos_to_compare] && gain[threadIdx_x] < gain[pos_to_compare])) {
        found[threadIdx_x] = found[pos_to_compare];
        gain[threadIdx_x] = gain[pos_to_compare];
        shared_read_index[threadIdx_x] = shared_read_index[pos_to_compare];
      }
    }
    __syncthreads();
  } 
}

__global__ void SyncBestSplitForLeafKernel(const int smaller_leaf_index, const int larger_leaf_index,
  const int* cuda_num_features,
  CUDASplitInfo* cuda_leaf_best_split_info,
  // input parameters
  const int* cuda_task_feature_index,
  const CUDASplitInfo* cuda_best_split_info,
  const uint32_t* cuda_feature_default_bins,
  const int num_tasks,
  const int num_tasks_aligned,
  const int num_blocks_per_leaf,
  const bool larger_only,
  const int num_leaves) {

  const uint32_t threadIdx_x = threadIdx.x;
  const uint32_t blockIdx_x = blockIdx.x;

  __shared__ bool best_found[NUM_TASKS_PER_SYNC_BLOCK];
  __shared__ double best_gain[NUM_TASKS_PER_SYNC_BLOCK];
  __shared__ uint32_t shared_read_index[NUM_TASKS_PER_SYNC_BLOCK];

  const bool is_smaller = (blockIdx_x < static_cast<unsigned int>(num_blocks_per_leaf) && !larger_only);
  const uint32_t leaf_block_index = (is_smaller || larger_only) ? blockIdx_x : (blockIdx_x - static_cast<unsigned int>(num_blocks_per_leaf));
  const int task_index = static_cast<int>(leaf_block_index * blockDim.x + threadIdx_x);
  const uint32_t read_index = is_smaller ? static_cast<uint32_t>(task_index) : static_cast<uint32_t>(task_index + num_tasks);
  if (task_index < num_tasks) {
    best_found[threadIdx_x] = cuda_best_split_info[read_index].is_valid;
    best_gain[threadIdx_x] = cuda_best_split_info[read_index].gain;
    shared_read_index[threadIdx_x] = read_index;
  } else {
    best_found[threadIdx_x] = false;
  }

  __syncthreads();
  ReduceBestSplit(best_found, best_gain, shared_read_index, NUM_TASKS_PER_SYNC_BLOCK);
  if (threadIdx.x == 0) {
    const int leaf_index_ref = is_smaller ? smaller_leaf_index : larger_leaf_index;
    const unsigned buffer_write_pos = static_cast<unsigned int>(leaf_index_ref) + leaf_block_index * num_leaves;
    const uint32_t best_read_index = shared_read_index[0];
    CUDASplitInfo* cuda_split_info = cuda_leaf_best_split_info + buffer_write_pos;
    const CUDASplitInfo* best_split_info = cuda_best_split_info + best_read_index;
    if (best_found[0]) {
      cuda_split_info->gain = best_gain[0];
      cuda_split_info->inner_feature_index = is_smaller ? cuda_task_feature_index[best_read_index] :
        cuda_task_feature_index[static_cast<int>(best_read_index) - num_tasks];
      cuda_split_info->default_left = best_split_info->default_left;
      cuda_split_info->threshold = best_split_info->threshold;
      cuda_split_info->left_sum_gradients = best_split_info->left_sum_gradients;
      cuda_split_info->left_sum_hessians = best_split_info->left_sum_hessians;
      cuda_split_info->left_count = best_split_info->left_count;
      cuda_split_info->left_gain = best_split_info->left_gain; 
      cuda_split_info->left_value = best_split_info->left_value;
      cuda_split_info->right_sum_gradients = best_split_info->right_sum_gradients;
      cuda_split_info->right_sum_hessians = best_split_info->right_sum_hessians;
      cuda_split_info->right_count = best_split_info->right_count;
      cuda_split_info->right_gain = best_split_info->right_gain; 
      cuda_split_info->right_value = best_split_info->right_value;
      cuda_split_info->is_valid = true;
    } else {
      cuda_split_info->gain = K_MIN_SCORE;
      cuda_split_info->is_valid = false;
    }
  }
}

__global__ void SyncBestSplitForLeafKernelAllBlocks(
  const int smaller_leaf_index,
  const int larger_leaf_index,
  const unsigned int num_blocks_per_leaf,
  const int num_leaves,
  CUDASplitInfo* cuda_leaf_best_split_info,
  const bool larger_only) {
  if (!larger_only) {
    if (blockIdx.x == 0) {
      for (unsigned int block_index = 1; block_index < num_blocks_per_leaf; ++block_index) {
        const unsigned int leaf_read_pos = static_cast<unsigned int>(smaller_leaf_index) + block_index * static_cast<unsigned int>(num_leaves);
        CUDASplitInfo* smaller_leaf_split_info = cuda_leaf_best_split_info + smaller_leaf_index;
        const CUDASplitInfo* other_split_info = cuda_leaf_best_split_info + leaf_read_pos;
        if ((other_split_info->is_valid && smaller_leaf_split_info->is_valid &&
          other_split_info->gain > smaller_leaf_split_info->gain) ||
            (!smaller_leaf_split_info->is_valid && other_split_info->is_valid)) {
            smaller_leaf_split_info->is_valid = other_split_info->is_valid;
            smaller_leaf_split_info->inner_feature_index = other_split_info->inner_feature_index;
            smaller_leaf_split_info->default_left = other_split_info->default_left;
            smaller_leaf_split_info->threshold = other_split_info->threshold;
            smaller_leaf_split_info->gain = other_split_info->gain;
            smaller_leaf_split_info->left_sum_gradients = other_split_info->left_sum_gradients;
            smaller_leaf_split_info->left_sum_hessians = other_split_info->left_sum_hessians;
            smaller_leaf_split_info->left_count = other_split_info->left_count;
            smaller_leaf_split_info->left_gain = other_split_info->left_gain;
            smaller_leaf_split_info->left_value = other_split_info->left_value;
            smaller_leaf_split_info->right_sum_gradients = other_split_info->right_sum_gradients;
            smaller_leaf_split_info->right_sum_hessians = other_split_info->right_sum_hessians;
            smaller_leaf_split_info->right_count = other_split_info->right_count;
            smaller_leaf_split_info->right_gain = other_split_info->right_gain;
            smaller_leaf_split_info->right_value = other_split_info->right_value;
        }
      }
    }
  }
  if (larger_leaf_index >= 0) {
    if (blockIdx.x == 1 || larger_only) {
      for (unsigned int block_index = 1; block_index < num_blocks_per_leaf; ++block_index) {
        const unsigned int leaf_read_pos = static_cast<unsigned int>(larger_leaf_index) + block_index * static_cast<unsigned int>(num_leaves);
        CUDASplitInfo* larger_leaf_split_info = cuda_leaf_best_split_info + larger_leaf_index;
        const CUDASplitInfo* other_split_info = cuda_leaf_best_split_info + leaf_read_pos;
        if ((other_split_info->is_valid && larger_leaf_split_info->is_valid &&
          other_split_info->gain > larger_leaf_split_info->gain) ||
            (!larger_leaf_split_info->is_valid && other_split_info->is_valid)) {
            larger_leaf_split_info->is_valid = other_split_info->is_valid;
            larger_leaf_split_info->inner_feature_index = other_split_info->inner_feature_index;
            larger_leaf_split_info->default_left = other_split_info->default_left;
            larger_leaf_split_info->threshold = other_split_info->threshold;
            larger_leaf_split_info->gain = other_split_info->gain;
            larger_leaf_split_info->left_sum_gradients = other_split_info->left_sum_gradients;
            larger_leaf_split_info->left_sum_hessians = other_split_info->left_sum_hessians;
            larger_leaf_split_info->left_count = other_split_info->left_count;
            larger_leaf_split_info->left_gain = other_split_info->left_gain;
            larger_leaf_split_info->left_value = other_split_info->left_value;
            larger_leaf_split_info->right_sum_gradients = other_split_info->right_sum_gradients;
            larger_leaf_split_info->right_sum_hessians = other_split_info->right_sum_hessians;
            larger_leaf_split_info->right_count = other_split_info->right_count;
            larger_leaf_split_info->right_gain = other_split_info->right_gain;
            larger_leaf_split_info->right_value = other_split_info->right_value;
        }
      }
    }
  }
}

void CUDABestSplitFinder::LaunchSyncBestSplitForLeafKernel(
  const int cpu_smaller_leaf_index,
  const int cpu_larger_leaf_index,
  const bool is_smaller_leaf_valid,
  const bool is_larger_leaf_valid) {

  int num_tasks = num_tasks_;
  int num_tasks_aligned = 1;
  num_tasks -= 1;
  while (num_tasks > 0) {
    num_tasks_aligned <<= 1;
    num_tasks >>= 1;
  }
  const int num_blocks_per_leaf = (num_tasks_ + NUM_TASKS_PER_SYNC_BLOCK - 1) / NUM_TASKS_PER_SYNC_BLOCK;
  if (cpu_larger_leaf_index >= 0 && is_smaller_leaf_valid && is_larger_leaf_valid) {
    SyncBestSplitForLeafKernel<<<num_blocks_per_leaf, NUM_TASKS_PER_SYNC_BLOCK, 0, cuda_streams_[0]>>>(
      cpu_smaller_leaf_index,
      cpu_larger_leaf_index,
      cuda_num_features_,
      cuda_leaf_best_split_info_,
      cuda_task_feature_index_,
      cuda_best_split_info_,
      cuda_feature_default_bins_,
      num_tasks_,
      num_tasks_aligned,
      num_blocks_per_leaf,
      false,
      num_leaves_);
    if (num_blocks_per_leaf > 1) {
      SyncBestSplitForLeafKernelAllBlocks<<<1, 1, 0, cuda_streams_[0]>>>(
        cpu_smaller_leaf_index,
        cpu_larger_leaf_index,
        num_blocks_per_leaf,
        num_leaves_,
        cuda_leaf_best_split_info_,
        false);
    }
    SynchronizeCUDADeviceOuter(__FILE__, __LINE__);
    SyncBestSplitForLeafKernel<<<num_blocks_per_leaf, NUM_TASKS_PER_SYNC_BLOCK, 0, cuda_streams_[1]>>>(
      cpu_smaller_leaf_index,
      cpu_larger_leaf_index,
      cuda_num_features_,
      cuda_leaf_best_split_info_,
      cuda_task_feature_index_,
      cuda_best_split_info_,
      cuda_feature_default_bins_,
      num_tasks_,
      num_tasks_aligned,
      num_blocks_per_leaf,
      true,
      num_leaves_);
    if (num_blocks_per_leaf > 1) {
      SyncBestSplitForLeafKernelAllBlocks<<<1, 1, 0, cuda_streams_[1]>>>(
        cpu_smaller_leaf_index,
        cpu_larger_leaf_index,
        num_blocks_per_leaf,
        num_leaves_,
        cuda_leaf_best_split_info_,
        true);
    }
  } else {
    const bool larger_only = (!is_smaller_leaf_valid && is_larger_leaf_valid);
    SyncBestSplitForLeafKernel<<<num_blocks_per_leaf, NUM_TASKS_PER_SYNC_BLOCK>>>(
      cpu_smaller_leaf_index,
      cpu_larger_leaf_index,
      cuda_num_features_,
      cuda_leaf_best_split_info_,
      cuda_task_feature_index_,
      cuda_best_split_info_,
      cuda_feature_default_bins_,
      num_tasks_,
      num_tasks_aligned,
      num_blocks_per_leaf,
      larger_only,
      num_leaves_);
    if (num_blocks_per_leaf > 1) {
      SynchronizeCUDADeviceOuter(__FILE__, __LINE__);
      SyncBestSplitForLeafKernelAllBlocks<<<1, 1>>>(
        cpu_smaller_leaf_index,
        cpu_larger_leaf_index,
        num_blocks_per_leaf,
        num_leaves_,
        cuda_leaf_best_split_info_,
        larger_only);
    }
  }
}

__global__ void FindBestFromAllSplitsKernel(const int* cuda_cur_num_leaves,
  int* out_best_leaf,
  const CUDASplitInfo* cuda_leaf_best_split_info,
  int* cuda_best_split_info_buffer) {
  const int cuda_cur_num_leaves_ref = *cuda_cur_num_leaves;
  __shared__ double thread_best_gain[NUM_THREADS_FIND_BEST_LEAF];
  __shared__ int thread_best_leaf[NUM_THREADS_FIND_BEST_LEAF];
  const unsigned int threadIdx_x = threadIdx.x;
  thread_best_gain[threadIdx_x] = K_MIN_SCORE;
  thread_best_leaf[threadIdx_x] = -1;
  const int num_leaves_per_thread = (cuda_cur_num_leaves_ref + NUM_THREADS_FIND_BEST_LEAF - 1) / NUM_THREADS_FIND_BEST_LEAF;
  const int cur_num_valid_threads = (cuda_cur_num_leaves_ref + num_leaves_per_thread - 1) / num_leaves_per_thread;
  if (threadIdx_x < static_cast<unsigned int>(cur_num_valid_threads)) {
    const int start = num_leaves_per_thread * threadIdx_x;
    const int end = min(start + num_leaves_per_thread, cuda_cur_num_leaves_ref);
    for (int leaf_index = threadIdx_x; leaf_index < cuda_cur_num_leaves_ref; leaf_index += cur_num_valid_threads) {
      const double leaf_best_gain = cuda_leaf_best_split_info[leaf_index].gain;
      if (cuda_leaf_best_split_info[leaf_index].is_valid && leaf_best_gain > thread_best_gain[threadIdx_x]) {
        thread_best_gain[threadIdx_x] = leaf_best_gain;
        thread_best_leaf[threadIdx_x] = leaf_index;
      }
    }
  }
  __syncthreads();
  ReduceBestGainForLeaves(thread_best_gain, thread_best_leaf, cur_num_valid_threads);
  if (threadIdx_x == 0) {
    const int best_leaf_index = thread_best_leaf[0];
    *out_best_leaf = best_leaf_index;
    //cuda_leaf_best_split_info[best_leaf_index].leaf_index = best_leaf_index;
    cuda_best_split_info_buffer[6] = thread_best_leaf[0];
  }
}

__global__ void PrepareLeafBestSplitInfo(const int smaller_leaf_index, const int larger_leaf_index,
  int* cuda_best_split_info_buffer,
  const CUDASplitInfo* cuda_leaf_best_split_info) {
  const unsigned int threadIdx_x = blockIdx.x;
  if (threadIdx_x == 0) {
    cuda_best_split_info_buffer[0] = cuda_leaf_best_split_info[smaller_leaf_index].inner_feature_index;
  } else if (threadIdx_x == 1) {
    cuda_best_split_info_buffer[1] = cuda_leaf_best_split_info[smaller_leaf_index].threshold;
  } else if (threadIdx_x == 2) {
    cuda_best_split_info_buffer[2] = cuda_leaf_best_split_info[smaller_leaf_index].default_left;
  }
  if (larger_leaf_index >= 0) { 
    if (threadIdx_x == 3) {
      cuda_best_split_info_buffer[3] = cuda_leaf_best_split_info[larger_leaf_index].inner_feature_index;
    } else if (threadIdx_x == 4) {
      cuda_best_split_info_buffer[4] = cuda_leaf_best_split_info[larger_leaf_index].threshold;
    } else if (threadIdx_x == 5) {
      cuda_best_split_info_buffer[5] = cuda_leaf_best_split_info[larger_leaf_index].default_left;
    }
  }
}

void CUDABestSplitFinder::LaunchFindBestFromAllSplitsKernel(const int* cuda_cur_num_leaves,
  const int smaller_leaf_index, const int larger_leaf_index, std::vector<int>* leaf_best_split_feature,
  std::vector<uint32_t>* leaf_best_split_threshold, std::vector<uint8_t>* leaf_best_split_default_left, int* best_leaf_index) {
  FindBestFromAllSplitsKernel<<<1, NUM_THREADS_FIND_BEST_LEAF, 0, cuda_streams_[1]>>>(cuda_cur_num_leaves, cuda_best_leaf_,
    cuda_leaf_best_split_info_,
    cuda_best_split_info_buffer_);
  PrepareLeafBestSplitInfo<<<6, 1, 0, cuda_streams_[0]>>>(smaller_leaf_index, larger_leaf_index,
    cuda_best_split_info_buffer_,
    cuda_leaf_best_split_info_);
  std::vector<int> cpu_leaf_best_split_info_buffer(7);
  SynchronizeCUDADeviceOuter(__FILE__, __LINE__);
  CopyFromCUDADeviceToHost<int>(cpu_leaf_best_split_info_buffer.data(), cuda_best_split_info_buffer_, 7);
  (*leaf_best_split_feature)[smaller_leaf_index] = cpu_leaf_best_split_info_buffer[0];
  (*leaf_best_split_threshold)[smaller_leaf_index] = static_cast<uint32_t>(cpu_leaf_best_split_info_buffer[1]);
  (*leaf_best_split_default_left)[smaller_leaf_index] = static_cast<uint8_t>(cpu_leaf_best_split_info_buffer[2]);
  if (larger_leaf_index >= 0) {
    (*leaf_best_split_feature)[larger_leaf_index] = cpu_leaf_best_split_info_buffer[3];
    (*leaf_best_split_threshold)[larger_leaf_index] = static_cast<uint32_t>(cpu_leaf_best_split_info_buffer[4]);
    (*leaf_best_split_default_left)[larger_leaf_index] = static_cast<uint8_t>(cpu_leaf_best_split_info_buffer[5]);
  }
  *best_leaf_index = cpu_leaf_best_split_info_buffer[6];
}

}  // namespace LightGBM

#endif  // USE_CUDA
