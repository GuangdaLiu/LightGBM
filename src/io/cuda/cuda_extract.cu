
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#include <LightGBM/dataset_loader.h>

namespace LightGBM {

__global__ void ValueToBinKernel(uint32_t* cuda_batch_bins_ptr[], double* cuda_bin_upper_bounds_ptr[], const int cuda_bin_upper_bounds_size[], const bool cuda_should_feature_mapped[], double* cuda_batch_value_ptr[], const data_size_t cur_cuda_batch_size) {
  data_size_t row_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int col_idx = blockIdx.y;
  if (row_idx < cur_cuda_batch_size && cuda_should_feature_mapped[col_idx]) {
    double value = cuda_batch_value_ptr[row_idx][col_idx];
    uint32_t bin = 0;
    // only consider NumericalBin now
    int l = 0;
    int r = cuda_bin_upper_bounds_size[col_idx];
    int m = 0;
    while (l < r) {
      m = (r + l - 1) / 2;
      if (value <= cuda_bin_upper_bounds_ptr[col_idx][m]) {
        r = m;
      } else {
        l = m + 1;
      }
    }
    bin = m;
    cuda_batch_bins_ptr[row_idx][col_idx] = bin;
  }
}

void DatasetLoader::LaunchValueToBinKernel(uint32_t* cuda_batch_bins_ptr[], double* cuda_bin_upper_bounds_ptr[], const int cuda_bin_upper_bounds_size[], const bool cuda_should_feature_mapped[], double* cuda_batch_value_ptr[], const data_size_t cur_cuda_batch_size, const int num_total_features) {
  const int num_threads_per_block = 1024;
  int num_blocks_for_row = (cur_cuda_batch_size + num_threads_per_block - 1) / num_threads_per_block;
  dim3 num_blocks(num_blocks_for_row, num_total_features);
  ValueToBinKernel<<<num_blocks, num_threads_per_block>>>(cuda_bin_upper_bounds_ptr, cuda_bin_upper_bounds_size, cuda_should_feature_mapped, cuda_batch_value_ptr, cur_cuda_batch_size);
}
  
}

#endif